#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include <optix_device.h>
#include "../../owl/include/owl/common/parallel/parallel_for.h"

using namespace owl;

__constant__ MyGlobals optixLaunchParams;

// bounding box programs 
template<typename SphereGeomType>
inline __device__ void boundsProg(const void *geomData,
                                  box3f &primBounds,
                                  const int primID)
{   
  const SphereGeomType &self = *(const SphereGeomType*)geomData;
  const Sphere sphere = self.data_pts[primID];
  primBounds = box3f().extend(sphere.center - self.rad)
		                  .extend(sphere.center + self.rad);	
  
}


OPTIX_BOUNDS_PROGRAM(Spheres)(const void  *geomData,
  box3f       &primBounds,
  const int    primID)
{ boundsProg<SpheresGeom>(geomData,primBounds,primID); }


// intersect program
OPTIX_INTERSECT_PROGRAM(Spheres)()
{ 
	
	const int primID = optixGetPrimitiveIndex();
	const SpheresGeom &selfs = owl::getProgramData<SpheresGeom>();
	Sphere self = selfs.data_pts[primID];
	const vec3f org = optixGetWorldRayOrigin();		

	float distance = 0.0;

#if (NORM == -1)  //dot
	distance = 1.00 - self.center.x * org.x
						- self.center.y * org.y
						- self.center.z * org.z;	
#elif (NORM == 0) //linfty
	double x  = std::abs(self.center.x - org.x);
	double y = std::abs(self.center.y - org.y);
	double z = abs(self.center.z - org.z);
	if(x > y )
		distance = x;
	else
		distance = y;
	if(distance < z)
		distance = z;
#elif (NORM > 0)
  	distance = std::pow(std::abs(self.center.x - org.x), NORM) 
			 + std::pow(std::abs(self.center.y - org.y), NORM)
			 + std::pow(std::abs(self.center.z - org.z), NORM);
#endif	

	if(distance < selfs.rad*selfs.rad){
    NeighGroup &param = owl::getPRD<NeighGroup>();
    int max_idx=0;
    for (int i = 1; i < KN; i++)
    {
      if (param.res[i].dist > param.res[max_idx].dist)
      max_idx = i;
    }
    if ( distance < param.res[max_idx].dist){
      param.res[max_idx].dist = distance;
      param.res[max_idx].ind = primID;
    }
  }
}	


OPTIX_RAYGEN_PROGRAM(rayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  int xID = optixGetLaunchIndex().x;
  NeighGroup param;
  for(int i=0; i<KN; i++){
    param.res[i].ind = -1;
    param.res[i].dist = FLOAT_MAX;
  }
  owl::Ray ray(self.query_pts[xID].center, vec3f(0,0,1), 0, 1.e-16f);
  owl::traceRay(self.world, ray, param);

  for(int i=0; i<KN; i++){
    optixLaunchParams.frameBuffer[xID*KN+i].ind = param.res[i].ind;
    optixLaunchParams.frameBuffer[xID*KN+i].dist = param.res[i].dist;
  }
}
