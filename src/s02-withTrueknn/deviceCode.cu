#include "hip/hip_runtime.h"
// All rights reserved by 
// Durga Keerthi Mandarapu, Vani Nagarajan, Artem Pelenitsyn, and Milind Kulkarni. 2024. 
// Arkade: k-Nearest Neighbor Search With Non-Euclidean Distances using GPU Ray Tracing. 

#include "deviceCode.h"
#include <optix_device.h>

using namespace owl;

__constant__ MyGlobals optixLaunchParams;

// bounding box program
template<typename SphereGeomType>
inline __device__ void boundsProg(const void *geomData,
                                  box3f &primBounds,
                                  const int primID)
{   
  const SphereGeomType &self = *(const SphereGeomType*)geomData;
  const Sphere sphere = self.data_pts[primID];
	primBounds = box3f()
		.extend(sphere.center - self.rad)
		.extend(sphere.center + self.rad);	
}


OPTIX_BOUNDS_PROGRAM(Spheres)(const void  *geomData,
  box3f       &primBounds,
  const int    primID)
{ boundsProg<SpheresGeom>(geomData,primBounds,primID); }

// intersect program
OPTIX_INTERSECT_PROGRAM(Spheres)()
{ 
	
	const int primID = optixGetPrimitiveIndex();
	const SpheresGeom &selfs = owl::getProgramData<SpheresGeom>();
	Sphere self = selfs.data_pts[primID];
	const vec3f org = optixGetWorldRayOrigin();		
  NeighGroup &param = owl::getPRD<NeighGroup>();
  
  // optixLaunchParams.intersections[qID]++;

  int check = 0;
  if(optixLaunchParams.round > 0 )
  {
    for (int i = 0; i < KN; i++) {
      if (param.res[i].ind == primID) {
        check = 1;
        break;
      }
    }
  }
	
  if (check == 0)
  {
    float distance = 0.0;
    #if (NORM == -1)  //dot
      distance = 1.00 - self.center.x * org.x
                - self.center.y * org.y
                - self.center.z * org.z;
    #elif (NORM == 0) //linfty
      double x  = std::abs(self.center.x - org.x);
      double y = std::abs(self.center.y - org.y);
      double z = abs(self.center.z - org.z);
      if(x > y )
        distance = x;
      else
        distance = y;
      if(distance < z)
        distance = z;
    #elif (NORM > 0)
        distance = std::pow(std::abs(self.center.x - org.x), NORM) 
          + std::pow(std::abs(self.center.y - org.y), NORM)
          + std::pow(std::abs(self.center.z - org.z), NORM);
    #endif	


    int max_idx=0;
    for (int i = 1; i < KN; i++)
    {
      if (param.res[i].dist > param.res[max_idx].dist)
        max_idx = i;
    }
    if (distance < param.res[max_idx].dist){
      param.res[max_idx].dist = distance;
      param.res[max_idx].ind = primID;
    }
  }

}	


OPTIX_RAYGEN_PROGRAM(rayGen)()
{
  
  int xID = optixGetLaunchIndex().x;
  
  if(optixLaunchParams.num_neighbors[xID] < KN)
  {
    const RayGenData &self = owl::getProgramData<RayGenData>();    
    NeighGroup param;
    for(int i=0; i<KN; i++)
    {
      param.res[i].ind = -1;
      param.res[i].dist = FLOAT_MAX;
    }
    owl::Ray ray(self.query_pts[xID].center, vec3f(0,0,1), 0, 1.e-16f);
    owl::traceRay(self.world, ray, param);

    int num_neighbors = 0; 
    for(int i=0; i<KN; i++){
      
      if (param.res[i].ind != -1)
      {
        num_neighbors++;
        optixLaunchParams.frameBuffer[xID*KN+i].ind = param.res[i].ind;
        optixLaunchParams.frameBuffer[xID*KN+i].dist = param.res[i].dist;
      }  
    }
    optixLaunchParams.num_neighbors[xID] = num_neighbors;
  }
  
}
